#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <vector>
#include <string>
#include <numeric>


#define CHECK_CUDA(func) {                                                      \
    hipError_t status = (func);                                                \
    if (status != hipSuccess) {                                                \
        std::cerr << "CUDA Error at line " << __LINE__ << ": "                  \
                  << hipGetErrorString(status) << std::endl;                   \
        exit(EXIT_FAILURE);                                                     \
    }                                                                           \
}

#define CHECK_CUBLAS(func) {                                                    \
    hipblasStatus_t status = (func);                                             \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                      \
        std::cerr << "cuBLAS Error at line " << __LINE__ << ": "                \
                  << status << std::endl;                                       \
        exit(EXIT_FAILURE);                                                     \
    }                                                                           \
}

#define CHECK_CUSPARSE(func) {                                                  \
    hipsparseStatus_t status = (func);                                           \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                    \
        std::cerr << "cuSPARSE Error at line " << __LINE__ << ": "              \
                  << status << std::endl;                                       \
        exit(EXIT_FAILURE);                                                     \
    }                                                                           \
}



void runSpmmTest(int A_ell_blocksize, int pattern){
    // -------------------------------------------------------------------
    // Parameters for matrices:
    constexpr int A_num_rows = 4096;
    constexpr int A_num_cols = 4096;
    constexpr int B_num_rows = 4096;
    constexpr int B_num_cols = 2048;
    constexpr int ldc = A_num_rows;       // leading dimension of C
    constexpr int ldb = B_num_rows;       // leading dimension of B
    const int C_size = ldc * B_num_cols;  // number of elements in result

    // -------------------------------------------------------------------
    // Create a sparse matrix A in blocked ELL format.
    // For a 4096x4096 matrix, there are 4096/4 = 1024 blocks per dimension.
    const int mb = A_num_rows / A_ell_blocksize;  // number of block rows (1024)
    const int nb = A_num_cols / A_ell_blocksize;  // number of block columns (1024)
    const int A_num_blocks = mb * nb;
    // For simplicity, we allocate storage for one block per dense block position.
    // In a typical ELL format, you might overallocate; here we assume each block is present.
    int *hA_columns = (int*)malloc(A_num_blocks * sizeof(int));
    for (int i = 0; i < mb; i++) {
        for (int j = 0; j < nb; j++) {
            hA_columns[i * nb + j] = j;
        }
    }
    // Each block is 4x4.
    __half *hA_values = (__half*)malloc(A_num_blocks * A_ell_blocksize * A_ell_blocksize * sizeof(__half));
    srand((unsigned)time(NULL));
    for (int i = 0; i < 4096 * 4096; i += pattern) {
        // Create an array with indices 0, 1, 2, 3.
        std::vector<int> indices(pattern);
        std::iota(indices.begin(), indices.end(), 0);
        // Shuffle indices using the Fisher–Yates algorithm.
        for (int j = 0; j < pattern; j++) {
            int rdx = j + rand() % (pattern - j);
            int temp = indices[j];
            indices[j] = indices[rdx];
            indices[rdx] = temp;
        }
        // Use the first two indices in the shuffled array as the positions for 1.
        int pos1 = indices[0];
        int pos2 = indices[1];
        
        // For each of the 4 positions, set to 1 if its index is pos1 or pos2, else 0.
        for (int j = 0; j < pattern; j++) {
            if (j == pos1 || j == pos2)
                hA_values[i + j] = __float2half(1.0f);
            else
                hA_values[i + j] = __float2half(0.0f);
        }
    }
    
    // Optional: Print the first 16 elements (4 groups of 4) to verify the pattern.
    //printf("First 16 elements (4 groups of 4):\n");
    //for (int i = 0; i < 32; i++) {
    //    printf("%8.2f ", __half2float(hA_values[i]));
    //}

    // Create dense matrix B with random values.
    const int B_size = ldb * B_num_cols;
    __half *hB = (__half*)malloc(B_size * sizeof(__half));
    for (int i = 0; i < B_size; i++) {
        hB[i] = __float2half(static_cast<float>(1));
    }
    
    // -------------------------------------------------------------------
    // Allocate device memory for the sparse matrix multiplication.
    int *dA_columns;
    __half *dA_values, *dB, *dC;
    CHECK_CUDA(hipMalloc((void**)&dA_columns, A_num_blocks * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&dA_values, A_num_blocks * A_ell_blocksize * A_ell_blocksize * sizeof(__half)));
    CHECK_CUDA(hipMalloc((void**)&dB, B_size * sizeof(__half)));
    CHECK_CUDA(hipMalloc((void**)&dC, C_size * sizeof(__half)));

    // Copy the host data to device.
    CHECK_CUDA(hipMemcpy(dA_columns, hA_columns, A_num_blocks * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dA_values, hA_values, A_num_blocks * A_ell_blocksize * A_ell_blocksize * sizeof(__half), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(dB, hB, B_size * sizeof(__half), hipMemcpyHostToDevice));
    // Initialize result matrix dC to zero.
    CHECK_CUDA(hipMemset(dC, 0, C_size * sizeof(__half)));

    // -------------------------------------------------------------------
    // Create cuSPARSE descriptors and perform the sparse SpMM.
    hipsparseHandle_t cusparseHandle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t matB, matC;
    void* dBuffer = NULL;
    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseCreate(&cusparseHandle));

    // Create the sparse matrix descriptor in blocked ELL format.
    CHECK_CUSPARSE(hipsparseCreateBlockedEll(&matA,
                                             A_num_rows, A_num_cols,
                                             A_ell_blocksize,
                                             A_num_cols,  // maximum blocks per row (here each row block is full)
                                             dA_columns,
                                             dA_values,
                                             HIPSPARSE_INDEX_32I,
                                             HIPSPARSE_INDEX_BASE_ZERO,
                                             HIP_R_16F));

    // Create dense matrix descriptors for B and C.
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, ldb, dB,
                                        HIP_R_16F, HIPSPARSE_ORDER_COL));
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matC, A_num_rows, B_num_cols, ldc, dC,
                                        HIP_R_16F, HIPSPARSE_ORDER_COL));

    // SpMM parameters.
    const float alpha = 1.0f;
    const float beta  = 0.0f;
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(cusparseHandle,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                           &alpha,
                                           matA, matB,
                                           &beta,
                                           matC,
                                           HIP_R_16F,
                                           HIPSPARSE_SPMM_ALG_DEFAULT,
                                           &bufferSize));
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
    //warm-up
    CHECK_CUSPARSE(hipsparseSpMM(cusparseHandle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha,
        matA, matB,
        &beta,
        matC,
        HIP_R_32F,
        HIPSPARSE_SPMM_ALG_DEFAULT,
        dBuffer));
    CHECK_CUDA(hipDeviceSynchronize()); // Ensure the warm-up is finished

    hipEvent_t startSpMM, stopSpMM;
    CHECK_CUDA(hipEventCreate(&startSpMM));
    CHECK_CUDA(hipEventCreate(&stopSpMM));
    CHECK_CUDA(hipEventRecord(startSpMM, 0));

    // Execute the sparse SpMM.
    CHECK_CUSPARSE(hipsparseSpMM(cusparseHandle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha,
                                matA, matB,
                                &beta,
                                matC,
                                HIP_R_32F,
                                HIPSPARSE_SPMM_ALG_DEFAULT,
                                dBuffer));

    // Clean up the cuSPARSE descriptors.
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matB));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matC));
    CHECK_CUSPARSE(hipsparseDestroy(cusparseHandle));

    CHECK_CUDA(hipEventRecord(stopSpMM, 0));
    CHECK_CUDA(hipEventSynchronize(stopSpMM));
    float spmmTime = 0.0f;
    CHECK_CUDA(hipEventElapsedTime(&spmmTime, startSpMM, stopSpMM));
    printf("Block size: %3d, Pattern: %3d, cuSPARSE SpMM execution time: %f ms\n", A_ell_blocksize, pattern, spmmTime);

    // Copy the sparse multiplication result from device to host.
    __half *hC_sparse = (__half*)malloc(C_size * sizeof(__half));
    CHECK_CUDA(hipMemcpy(hC_sparse, dC, C_size * sizeof(__half), hipMemcpyDeviceToHost));
    //printf("First 16 elements (4 groups of 4):\n");
    //for (int i = 0; i < 32; i++) {
    //    printf("%8.2f ", __half2float(hC_sparse[i]));
    //}
    if(A_ell_blocksize == 1024 && pattern == 1024){
    // Allocate device memory for the dense version of A and for the reference result.
    __half *dA_dense, *dC_dense;
    CHECK_CUDA(hipMalloc(&dA_dense, A_num_rows * A_num_cols * sizeof(__half)));
    CHECK_CUDA(hipMalloc(&dC_dense, C_size * sizeof(__half)));

    // Copy the dense A from host to device.
    CHECK_CUDA(hipMemcpy(dA_dense, hA_values, A_num_rows * A_num_cols * sizeof(__half), hipMemcpyHostToDevice));
    // Initialize dC_dense to zero.
    CHECK_CUDA(hipMemset(dC_dense, 0, C_size * sizeof(__half)));

    // Create a cuBLAS handle.
    hipblasHandle_t cublas_handle;
    CHECK_CUBLAS(hipblasCreate(&cublas_handle));
    CHECK_CUBLAS(hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH));
    __half h_alpha = __float2half(1.0f);
    __half h_beta  = __float2half(0.0f);
    CHECK_CUBLAS(hipblasGemmEx(cublas_handle,
        HIPBLAS_OP_T, HIPBLAS_OP_N,
        A_num_rows, B_num_cols, A_num_cols,
        &h_alpha,
        dA_dense, HIP_R_16F, A_num_cols,
        dB,       HIP_R_16F, A_num_cols,
        &h_beta,
        dC_dense, HIP_R_16F, A_num_rows,
        HIP_R_16F,
        HIPBLAS_GEMM_DEFAULT));
    CHECK_CUDA(hipDeviceSynchronize()); 
    // Perform dense GEMM: dC_dense = dA_dense * dB.
    hipEvent_t startDense, stopDense;
    CHECK_CUDA(hipEventCreate(&startDense));
    CHECK_CUDA(hipEventCreate(&stopDense));
    CHECK_CUDA(hipEventRecord(startDense));
    CHECK_CUBLAS(hipblasGemmEx(cublas_handle,
        HIPBLAS_OP_T, HIPBLAS_OP_N,
                              A_num_rows, B_num_cols, A_num_cols,
                              &h_alpha,
                              dA_dense, HIP_R_16F, A_num_cols,
                              dB, HIP_R_16F, A_num_cols,
                              &h_beta,
                              dC_dense, HIP_R_16F, A_num_rows,
                              HIP_R_16F,
                              HIPBLAS_GEMM_DEFAULT));
    CHECK_CUDA(hipEventRecord(stopDense));
    CHECK_CUDA(hipEventSynchronize(stopDense));
    float denseTime = 0.0f;
    CHECK_CUDA(hipEventElapsedTime(&denseTime, startDense, stopDense));
   printf("Dense GEMM execution time for reference: %f ms\n", denseTime);

    // Copy the dense result from device to host.
    __half *hC_dense = (__half*)malloc(C_size * sizeof(__half));
    CHECK_CUDA(hipMemcpy(hC_dense, dC_dense, C_size * sizeof(__half), hipMemcpyDeviceToHost));


    free(hC_dense);

    CHECK_CUDA(hipFree(dA_dense));
    CHECK_CUDA(hipFree(dC_dense));

    CHECK_CUBLAS(hipblasDestroy(cublas_handle));
    CHECK_CUDA(hipEventDestroy(startDense));
    CHECK_CUDA(hipEventDestroy(stopDense));
}
    // printf("First 16 elements (4 groups of 4):\n");
    // for (int i = 0; i < 32; i++) {
    //     printf("%8.2f ", __half2float(hC_dense[i]));
    // }
    // -------------------------------------------------------------------
    // Clean up all allocated resources.
    free(hA_columns);
    free(hA_values);
    free(hB);
    free(hC_sparse);
    

    CHECK_CUDA(hipFree(dBuffer));
    CHECK_CUDA(hipFree(dA_columns));
    CHECK_CUDA(hipFree(dA_values));
    CHECK_CUDA(hipFree(dB));
    CHECK_CUDA(hipFree(dC));
    

    

    CHECK_CUDA(hipEventDestroy(startSpMM));
    CHECK_CUDA(hipEventDestroy(stopSpMM));
    

}



int main(){
    int blockSizes[] = {4, 8, 16, 32, 64, 128, 256, 512, 1024};
    int patternSizes[] = {4, 8, 16, 32, 64, 128, 256, 512, 1024};
    //runSpmmTest(4,64);
    
    
     for(int i = 0; i < 9; i++){
         for(int j = 0; j < 9; j++){
             runSpmmTest(blockSizes[i], patternSizes[j]);
         }
     }
    return 0;
}
